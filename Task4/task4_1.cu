#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include "stdio.h"
#include "hip/hip_runtime.h"
#include ""

using namespace std;

const int N = 4;
const int M = 4;
const int thx = 2;
const int thy = 2;


__global__ void add(int* a, int* b, int* c){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int ind = i * N + j;
	c[ind] = a[ind] + b[ind];
}

int main() {
	int* a, * b, * c;
	hipMallocManaged(&a, M * N * sizeof(int));
	hipMallocManaged(&b, M * N * sizeof(int));
	hipMallocManaged(&c, M * N * sizeof(int));

	for (int i = 0; i < M * N; i++){
		a[i] = i;
		b[i] = i*i;
	}

	dim3 blocks(N / thx, M / thy);
	dim3 threads(thx, thy);

	add << < blocks, threads >> > (a, b, c);
	hipDeviceSynchronize();

	for (int i = 0; i < N; i++){
		for (int j = 0; j < M; j++)
			cout << c[i * N + j] << endl;
		cout << endl;
	}

	hipFree(a);
	hipFree(b);
	hipFree(c);
}
