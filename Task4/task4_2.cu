#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include "stdio.h"
#include "hip/hip_runtime.h"
#include ""

using namespace std;
typedef struct {
	int width;
	int height;
	int step;
	float* m;
} mat;

#define get_elem(A, r, c) (A.m[r * A.step + c])
#define set_elem(A, r, c, val) A.m[r * A.step + c] = val
#define BLOCK_SIZE 2

__device__ mat get_submat(mat A, int row, int col){
	mat Asub;
	Asub.height = BLOCK_SIZE;
	Asub.width = BLOCK_SIZE;
	Asub.step = A.step;
	Asub.m = &A.m[A.step * BLOCK_SIZE * row + BLOCK_SIZE * col];
	return Asub;
}

__global__ void matmul(mat A, mat B, mat C){
	int blockCol = blockIdx.x;
	int blockRow = blockIdx.y;

	mat Csub = get_submat(C, blockRow, blockCol);

	int row = threadIdx.y;
	int col = threadIdx.x;
	float cij = 0;
	for (int m = 0; m < (A.height / BLOCK_SIZE); ++m) {
		mat Asub = get_submat(A, blockRow, m);

		mat Bsub = get_submat(B, m, blockCol);

		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		As[row][col] = get_elem(Asub, row, col);
		Bs[row][col] = get_elem(Bsub, row, col);

		__syncthreads();
		for (int e = 0; e < BLOCK_SIZE; ++e)
			cij += As[row][e] * Bs[e][col];

		__syncthreads();
	}

	set_elem(Csub, row, col, cij);
}

int main(){
	float m1[] = { 1,2,3,4,
				5,6,7,8,
				9,10,11,12,
				13,14,15,16 };

	float m2[] = { 16,15,14,13,
				12,11,10,9,
				8,7,6,5,
				4,3,2,1 };

	float* m3 = (float*)malloc(4 * 4 * sizeof(float));

	mat A = { .width = 4, .height = 4, .step = 4, .m = m1 };
	mat B = { .width = 4, .height = 4, .step = 4, .m = m2 };
	mat C = { .width = 4, .height = 4, .step = 4, .m = m3 };

	mat d_A = { .width = A.width, .height = A.height, .step = A.height };
	int size = A.height * A.width * sizeof(float);
	hipMalloc(&d_A.m, size);
	hipMemcpy(d_A.m, A.m, size, hipMemcpyHostToDevice);

	mat d_B = { .width = B.width, .height = B.height, .step = B.height };
	size = B.height * B.width * sizeof(float);
	hipMalloc(&d_B.m, size);
	hipMemcpy(d_B.m, B.m, size, hipMemcpyHostToDevice);

	mat d_C = { .width = C.width, .height = C.height, .step = C.height };
	size = C.height * C.width * sizeof(float);
	hipMalloc(&d_C.m, size);

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.height / dimBlock.x, A.width / dimBlock.y);
	matmul << <dimGrid, dimBlock >> > (d_A, d_B, d_C);

	hipMemcpy(C.m, d_C.m, size, hipMemcpyDeviceToHost);

	int N = C.width, M = C.height;
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < M; j++) {
			float elem = C.m[i * C.step + j];
			cout << elem << " ";
		}
		cout << endl;
	}

	hipFree(d_A.m);
	hipFree(d_B.m);
	hipFree(d_C.m);
	free(m3);
}
